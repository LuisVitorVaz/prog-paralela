#include <iostream>
#include <hip/hip_runtime.h>

const int N = 1000; // Tamanho dos vetores
__device__ int somaValores2(int a, int b){
    printf("estou dentro da gpu :\n");
    return a+b;
}
__device__ int mult(int a, int b){
    return a*b;
}
// Kernel para a soma de dois vetores
__global__ void somaVetores(const int *a, const int *b, int *resultado) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        resultado[i] = somaValores2(a[i],  b[i]);
        resultado[i] = mult(a[i],  resultado[i]);
    }
}

int main() {
    // Inicializa os vetores no host
    int vetorA[N], vetorB[N], resultado[N];

    for (int i = 0; i < N; ++i) {
        vetorA[i] = i;
        vetorB[i] = 2 * i;
    }

    // Aloca memória na GPU
    int *d_vetorA, *d_vetorB, *d_resultado;
    hipMalloc((void**)&d_vetorA, sizeof(int) * N);
    hipMalloc((void**)&d_vetorB, sizeof(int) * N);
    hipMalloc((void**)&d_resultado, sizeof(int) * N);

    // Copia os dados do host para a GPU
    hipMemcpy(d_vetorA, vetorA, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_vetorB, vetorB, sizeof(int) * N, hipMemcpyHostToDevice);

    // Define a configuração do lançamento do kernel
    int threadsPorBloco = 256;
    int blocos = (N + threadsPorBloco - 1) / threadsPorBloco;

    // Chama o kernel para a soma dos vetores
    somaVetores<<<blocos, threadsPorBloco>>>(d_vetorA, d_vetorB, d_resultado);
        
    // Copia o resultado de volta para o host
    hipMemcpy(resultado, d_resultado, sizeof(int) * N, hipMemcpyDeviceToHost);

    // Exibe alguns resultados
    std::cout << "Alguns resultados da soma dos vetores:\n";
    for (int i = 0; i < N; ++i) {
        std::cout << resultado[i] << " ";
    }
    std::cout << std::endl;

    // Libera memória na GPU
    hipFree(d_vetorA);
    hipFree(d_vetorB);
    hipFree(d_resultado);

    return 0;
}
